#include "hip/hip_runtime.h"
#include "book.h"

#define imin(a, b) (a < b ? a : b)
#define sum_squares(x) (x * (x + 1) * (2 * x + 1) / 6)

constexpr int N = 33 * 1024;
constexpr int threadsPerBlock = 1024;

__global__ void dot (float* a, float* b, float* c) {
    // The buffer of shared memory.
    __shared__ float cache[threadsPerBlock];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N) {
        temp = a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;

    __syncthreads ();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads ();
        i /= 2;
    }
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

constexpr int blocksPerGrid = imin (32, (N + threadsPerBlock - 1) / threadsPerBlock);

int main () {
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    a = new float[N];
    b = new float[N];
    partial_c = new float[N];

    HANDLE_ERROR (hipMalloc ((void**)&dev_a, sizeof (float) * N));
    HANDLE_ERROR (hipMalloc ((void**)&dev_b, sizeof (float) * N));
    HANDLE_ERROR (hipMalloc ((void**)&dev_partial_c, sizeof (float) * blocksPerGrid));

    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    HANDLE_ERROR (hipMemcpy (dev_a, a, N * sizeof (float), hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_b, b, N * sizeof (float), hipMemcpyHostToDevice));

    dot<<<blocksPerGrid, threadsPerBlock>>> (dev_a, dev_b, dev_partial_c);

    HANDLE_ERROR (hipMemcpy (partial_c, dev_partial_c, blocksPerGrid * sizeof (float),
                              hipMemcpyDeviceToHost));
    
    // finish up on the CPU side  
    c = 0;  
    for (int i=0; i<blocksPerGrid; i++) {  
        c += partial_c[i];  
    }

    printf ("Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares ((float)(N - 1))); // free memory on the GPU side
    hipFree (dev_a);
    hipFree (dev_b);
    hipFree (dev_partial_c); // free memory on the CPU side
    delete[] a;
    delete[] b;
    delete[] partial_c;
    return 0;
}