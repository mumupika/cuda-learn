#include "hip/hip_runtime.h"
#include "book.h"

#define N 10

__global__ void add (int* a, int* b, int* c) {
    int tid = threadIdx.x;
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main () {
    int a[N], b[N], c[N];
    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    int *dev_a, *dev_b, *dev_c;
    HANDLE_ERROR (hipMalloc ((void**)&dev_a, sizeof (int) * N));
    HANDLE_ERROR (hipMalloc ((void**)&dev_b, sizeof (int) * N));
    HANDLE_ERROR (hipMalloc ((void**)&dev_c, sizeof (int) * N));

    HANDLE_ERROR (hipMemcpy (dev_a, a, sizeof (int) * N, hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_b, b, sizeof (int) * N, hipMemcpyHostToDevice));

    add<<<1, N>>> (dev_a, dev_b, dev_c);

    HANDLE_ERROR (hipMemcpy (c, dev_c, sizeof (int) * N, hipMemcpyDeviceToHost));
    for (int i = 0; i < N; i++) {
        printf ("%d + %d = %d\n", a[i], b[i], c[i]);
    }
    return 0;
}