#include "hip/hip_runtime.h"
#include "book.h"

#define N (33 * 1024)

__global__ void add (int* a, int* b, int* c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main () {
    int a[N], b[N], c[N];
    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    int *dev_a, *dev_b, *dev_c;
    HANDLE_ERROR (hipMalloc ((void**)&dev_a, sizeof (int) * N));
    HANDLE_ERROR (hipMalloc ((void**)&dev_b, sizeof (int) * N));
    HANDLE_ERROR (hipMalloc ((void**)&dev_c, sizeof (int) * N));

    HANDLE_ERROR (hipMemcpy (dev_a, a, sizeof (int) * N, hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_b, b, sizeof (int) * N, hipMemcpyHostToDevice));

    constexpr int UNIT = 128;
    add<<<UNIT, UNIT>>> (dev_a, dev_b, dev_c);

    HANDLE_ERROR (hipMemcpy (c, dev_c, sizeof (int) * N, hipMemcpyDeviceToHost));
    for (int i = 0; i < N; i++) {
        printf ("%d + %d = %d\n", a[i], b[i], c[i]);
    }
    return 0;
}