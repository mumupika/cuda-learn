/*
 *  Ref: https://docs.nvidia.com/deeplearning/nccl/user-guide/docs/examples.html
 *  Compile: nvcc  -lnccl -ccbin g++ -std=c++11 -O3 -g one_devices_per_thread.cu.cu -o one_devices_per_thread
 */

#include "comm.h"
#include <pthread.h>

pthread_mutex_t mutex;

ncclUniqueId id;

void *thread_function(void *arg)
{
    int size = 32 * 1024 * 1024;
    int gpu_id = *(int *)arg;
    hipSetDevice(gpu_id);

    ncclComm_t comm;
    NCCLCHECK(ncclCommInitRank(&comm, my_nranks, id, gpu_id));

    float *sendbuff;
    float *recvbuff;
    float *hostData;
    hipStream_t s;

    hostData = (float *)malloc(size * sizeof(float));
    for (int i = 0; i < size; ++i) {
        hostData[i] = float(i);
    }

    CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
    CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
    hipMemcpy(sendbuff, hostData, size * sizeof(float), hipMemcpyHostToDevice);
    CUDACHECK(hipStreamCreate(&s));

    NCCLCHECK(ncclAllReduce((const void *)sendbuff, (void *)recvbuff, size, ncclFloat, ncclSum, comm, s));

    // completing NCCL operation by synchronizing on the CUDA stream
    CUDACHECK(hipStreamSynchronize(s));
    ncclCommDestroy(comm);

    hipMemcpy(hostData, recvbuff, size * sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < 1000; i++) {
        pthread_mutex_lock(&mutex);
        printf("GPU:%d data: %f.\n", gpu_id, hostData[i]);
        pthread_mutex_unlock(&mutex);
    }

    CUDACHECK(hipFree(sendbuff));
    CUDACHECK(hipFree(recvbuff));
    free(hostData);

    return NULL;
}

int main(int argc, char *argv[])
{
    pthread_mutex_init(&mutex, NULL);
    env_init(argc, argv);
    pthread_t threads[8];
    NCCLCHECK(ncclGetUniqueId(&id));
    for (int i = 0; i < my_nranks; ++i) {
        int *id_pointer = &gpu_ids[i];
        pthread_create(&threads[i], NULL, thread_function, id_pointer);
    }

    for (int i = 0; i < my_nranks; ++i) {
        pthread_join(threads[i], NULL);
    }

    printf("Finished successfully.\n");
    pthread_mutex_unlock(&mutex);
    return 0;
}