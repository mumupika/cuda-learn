#include "hip/hip_runtime.h"
/*
    For fault tolerance.

    Asynchoronously submit task -> wait until timeout -> broadcast status -> restart.
*/

#include "comm.h"
#include "mpi.h"
#include "mpi_comm.h"
#include <chrono>
#include <nccl.h>

bool check_timeout (const std::chrono::time_point<std::chrono::steady_clock>& start,
                    int timeout) {
    const auto end = std::chrono::steady_clock::now ();
    if (std::chrono::duration_cast<std::chrono::seconds> (end - start) <
        std::chrono::seconds (timeout)) {
        return false;
    }
    return true;
}

ncclResult_t restartNCCL (ncclComm_t* comm, ncclUniqueId* id, int myRank, int nRanks) {
    // finalizing NCCL
    ncclCommDestroy (*comm);

    // Restart again.
    if (myRank == 0) {
        ncclGetUniqueId (id);
    }
    MPICHECK (MPI_Bcast ((void*)&id, sizeof (ncclUniqueId), MPI_BYTE, 0, MPI_COMM_WORLD));

    // Trying to restart init again. If failed, report failed and exit.
    ncclConfig_t config = NCCL_CONFIG_INITIALIZER;
    ncclResult_t async_state = ncclInProgress;
    const auto start = std::chrono::steady_clock::now ();
    int timeout = 30;
    config.blocking = 0;
    NCCLCHECK (ncclGroupStart ());
    NCCLCHECK (ncclCommInitRankConfig (comm, nRanks, *id, myRank, &config));
    do {
        NCCLCHECK (ncclCommGetAsyncError (*comm, &async_state));
    } while (async_state == ncclInProgress && check_timeout (start, timeout) != true);
    NCCLCHECK (ncclGroupEnd ());
    if (check_timeout (start, timeout) == true || async_state != ncclSuccess) {
        ncclCommAbort (*comm);
        return async_state;
    }
    return async_state; // ncclSuccess can be return.
}

void recover_data (float* dev_s, float* dev_r, float* host, int size) {
    CUDACHECK (hipMemset (dev_r, 0, sizeof (float) * size));
    CUDACHECK (hipMemcpy (dev_s, host, sizeof (float) * size, hipMemcpyHostToDevice));
}

void reportErrorGlobally (bool abortFlag, bool *globalFlag, int myRank) {
    abortFlag = true;
    *globalFlag = abortFlag;
    MPI_Bcast ((void*)globalFlag, sizeof (globalFlag), MPI_BYTE, myRank, MPI_COMM_WORLD);
}

void fault_tolearance_all_reduce (int myRank, int nRanks, int localRank) {
    int size = 32 * 1024 * 1024;

    ncclUniqueId id;
    ncclComm_t comm;
    float *sendbuff, *recvbuff, *hostbuff;
    hipStream_t s;

    // =================== ALLOCATE BASIC RESOURCES ===================

    // get NCCL unique ID at rank 0 and broadcast it to all others
    if (myRank == 0)
        ncclGetUniqueId (&id);
    MPICHECK (MPI_Bcast ((void*)&id, sizeof (id), MPI_BYTE, 0, MPI_COMM_WORLD));

    // picking a GPU based on localRank, allocate device buffers
    CUDACHECK (hipSetDevice (localRank));
    CUDACHECK (hipMalloc (&sendbuff, size * sizeof (float)));
    CUDACHECK (hipMalloc (&recvbuff, size * sizeof (float)));
    hostbuff = (float*)malloc (sizeof (float) * size);
    for (int i = 1.0; i < size; i++) {
        hostbuff[i] = float (i);
    }
    CUDACHECK (hipMemcpy (sendbuff, hostbuff, sizeof (float) * size, hipMemcpyHostToDevice));
    CUDACHECK (hipStreamCreate (&s));

    // =================== Start Initialize comm ===================

    // Using clock and flag to record status of Initialization with fault tolerance.
    bool globalFlag = true;
    bool abortFlag = false;
    ncclConfig_t config = NCCL_CONFIG_INITIALIZER;
    ncclResult_t async_state = ncclInProgress;
    int timeout = 30;
    config.blocking = 0;

    // FIXME: One device per process can not use GroupStart/GroupEnd?
    NCCLCHECK (ncclGroupStart ());
    auto start = std::chrono::steady_clock::now ();
    NCCLCHECK (ncclCommInitRankConfig (&comm, nRanks, id, myRank, &config));
    do {
        NCCLCHECK (ncclCommGetAsyncError (comm, &async_state));
    } while (async_state == ncclInProgress && check_timeout (start, timeout) != true);
    NCCLCHECK (ncclGroupEnd ());

    // Broadcast to the whole processes. We need a barrier since here has a divergent.
    if (check_timeout (start, timeout) == true || async_state != ncclSuccess) {
        reportErrorGlobally (abortFlag, &globalFlag, myRank);
    }
    MPI_Barrier (MPI_COMM_WORLD);
    if (globalFlag == true) {
        ncclCommAbort (comm);
        // Free all resources and Renew again!
        NCCLCHECK (restartNCCL (&comm, &id, myRank, nRanks));
    }

    // =================== Start all reduce ===================

    globalFlag = true;
    abortFlag = false;
    NCCLCHECK (ncclGroupStart ());
    start = std::chrono::steady_clock::now ();
    NCCLCHECK (ncclAllReduce ((const void*)sendbuff, (void*)recvbuff, size,
                              ncclFloat, ncclSum, comm, s));
    NCCLCHECK (ncclGroupStart ());
    do {
        NCCLCHECK (ncclCommGetAsyncError (comm, &async_state));
    } while (async_state != ncclSuccess && check_timeout (start, timeout) != true);
    // Broadcast to the whole processes. We need a barrier since here has a divergent.
    if (check_timeout (start, timeout) == true || async_state != ncclSuccess) {
        reportErrorGlobally (abortFlag, &globalFlag, myRank);
    }
    MPI_Barrier (MPI_COMM_WORLD);
    if (globalFlag == true) {
        ncclCommAbort (comm);
        // Free all resources and Renew again!
        NCCLCHECK (restartNCCL (&comm, &id, myRank, nRanks));

        // Reset the data and Retry again.
        recover_data (sendbuff, recvbuff, hostbuff, size);
        globalFlag = true;
        abortFlag = false;
        NCCLCHECK (ncclGroupStart ());
        start = std::chrono::steady_clock::now ();
        NCCLCHECK (ncclAllReduce ((const void*)sendbuff, (void*)recvbuff, size,
                                  ncclFloat, ncclSum, comm, s));
        NCCLCHECK (ncclGroupStart ());
        do {
            NCCLCHECK (ncclCommGetAsyncError (comm, &async_state));
        } while (async_state != ncclSuccess && check_timeout (start, timeout) != true);
        NCCLCHECK (async_state); // Check the async_state and exit immediately if failed.
    }

    // completing NCCL operation by synchronizing on the CUDA stream
    CUDACHECK (hipStreamSynchronize (s));

    // free device buffers
    CUDACHECK(hipMemcpy(hostbuff, recvbuff, sizeof(float) * size, hipMemcpyDeviceToHost));
    CUDACHECK (hipFree (sendbuff));
    CUDACHECK (hipFree (recvbuff));

    // finalizing NCCL
    ncclCommDestroy (comm);

    // finalizing MPI
    MPICHECK (MPI_Finalize ());

    printf ("[MPI Rank %d] Success \n", myRank);
    return;
}

int main (int argc, char* argv[]) {

    int myRank, nRanks, localRank = 0;

    // initializing MPI
    MPICHECK (MPI_Init (&argc, &argv));
    MPICHECK (MPI_Comm_rank (MPI_COMM_WORLD, &myRank)); // MyRank -> stands the process rank in mpi.
    MPICHECK (MPI_Comm_size (MPI_COMM_WORLD, &nRanks)); // How many processes used in openmpi.

    // calculating localRank based on hostname which is used in selecting a GPU
    uint64_t* hostHashs = (uint64_t*)malloc (sizeof (uint64_t) * nRanks);
    char hostname[1024];
    getHostName (hostname, 1024);
    hostHashs[myRank] = getHostHash (hostname);
    MPICHECK (MPI_Allgather (MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs,
                             sizeof (uint64_t), MPI_BYTE, MPI_COMM_WORLD));
    for (int p = 0; p < nRanks; p++) {
        if (p == myRank)
            break;
        if (hostHashs[p] == hostHashs[myRank])
            localRank++;
    }

    return 0;
}