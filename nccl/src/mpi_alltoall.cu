#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <nccl.h>
#include <stdint.h>

#include "comm.h"
#include "mpi_comm.h"

/**
Assuming we have:

GPU 0: 0,1,2,3,4,5,6,7
GPU 1: 8,9 10,11,12,13,14,15
GPU 2: 16,17,18,19,20,21,22,23
GPU 3: 24,25,26,27,28,29,30,31

After alltoall we should have:

GPU 0: 0,1,8,9,16,17,24,25
GPU 1: 2,3,10,11,18,19,26,27
GPU 2: 4,5,12,13,20,21,28,29
GPU 3: 6,7,14,15,22,23,30,31

Requires: if k ranks -> buffersize should be k * N.
*/

__global__ void fill_data (int dev_rank, int size, float* buff) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        buff[tid] = tid + dev_rank * size;
    }
    __syncthreads ();
}

void non_blocking_all_to_all (int myRank, int nRanks, int localRank) {
    // Set up the communication area and broadcast to all devices to host.
    ncclUniqueId id;
    if (myRank == 0) {
        NCCLCHECK (ncclGetUniqueId (&id));
    }
    MPICHECK (MPI_Bcast ((void*)&id, sizeof (ncclUniqueId), MPI_BYTE, 0, MPI_COMM_WORLD));

    int nDev = 2; // 1 Process -> 2 GPU

    float** sendbuff = (float**)malloc (nDev * sizeof (float*));
    float** recvbuff = (float**)malloc (nDev * sizeof (float*));
    hipStream_t* s = (hipStream_t*)malloc (nDev * sizeof (hipStream_t));
    int size = 8;

    for (int i = 0; i < nDev; i++) {
        CUDACHECK (hipSetDevice (localRank * nDev + i));
        CUDACHECK (hipMalloc ((void**)&sendbuff[i], sizeof (float) * size));
        CUDACHECK (hipMalloc ((void**)&recvbuff[i], sizeof (float) * size));
        // fill in the data.
        int threadsPerBlock = 1024;
        int blockPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        CUDACHECK (hipStreamCreate (&s[i]));
        fill_data<<<blockPerGrid, threadsPerBlock, 0, s[i]>>> (myRank * nDev + i,
                                                               size, sendbuff[i]);
        CUDACHECK (hipMemset (recvbuff[i], 0, sizeof (float) * size));
    }

    // Synchronize until all stream completed...
    for (int i = 0; i < nDev; i++) {
        CUDACHECK (hipStreamSynchronize (s[i]));
    }

    // Display the data out.
    for (int i = 0; i < nDev; i++) {
        printf ("[Rank %d]: ", myRank * nDev + i);
        float* tempBuff = (float*)malloc (sizeof (float) * size);
        hipMemcpy (tempBuff, sendbuff[i], sizeof (float) * size, hipMemcpyDeviceToHost);
        for (int j = 0; j < size; j++) {
            printf ("%f ", tempBuff[j]);
        }
        printf ("\n");
        free (tempBuff);
    }

    // Initializing NCCL ranks. Group initialize the comms.
    ncclComm_t* comms = (ncclComm_t*)malloc (sizeof (ncclComm_t) * nDev);
    ncclResult_t* state = (ncclResult_t*)malloc (sizeof (ncclResult_t) * nDev);
    ncclConfig_t* config = (ncclConfig_t*)malloc (sizeof (ncclConfig_t) * nDev);
    ncclResult_t ret = ncclInProgress;
    for (int i = 0; i < nDev; i++) {
        config[i] = NCCL_CONFIG_INITIALIZER;
        config[i].blocking = 0;
    }

    ncclGroupStart ();
    for (int i = 0; i < nDev; i++) {
        CUDACHECK (hipSetDevice (localRank * nDev + i));
        ncclCommInitRankConfig (&comms[i], nRanks * nDev, id, myRank * nDev + i,
                                &config[i]);
    }
    ret = ncclGroupEnd ();
    if (ret == ncclInProgress) {
        for (int i = 0; i < nDev; i++) {
            // Wait until one complete. Then another.
            do {
                NCCLCHECK (ncclCommGetAsyncError (comms[i], &state[i]));
            } while (state[i] == ncclInProgress);
            NCCLCHECK (state[i]);
        }
    } else if (ret == ncclSuccess) {
        printf ("NCCL kernel issue succeeded\n");
    } else {
        // Error occurred.
        NCCLCHECK (ret);
    }

    // NONBLOCKING allReduce.
    ncclGroupStart ();
    for (int i = 0; i < nDev; i++) {
        ncclAlltoAll (reinterpret_cast<const char*> (sendbuff[i]),
                      static_cast<void*> (recvbuff[i]), size / (nRanks * nDev),
                      ncclFloat, comms[i], s[i]);
    }
    ret = ncclGroupEnd ();
    if (ret == ncclInProgress) {
        for (int i = 0; i < nDev; i++) {
            // Wait until one complete. Then another.
            do {
                NCCLCHECK (ncclCommGetAsyncError (comms[i], &state[i]));
            } while (state[i] == ncclInProgress);
            NCCLCHECK (state[i]);
        }
    } else if (ret == ncclSuccess) {
        printf ("NCCL kernel issue succeeded\n");
    } else {
        // Error occurred.
        NCCLCHECK (ret);
    }

    for (int i = 0; i < nDev; i++) {
        hipStreamSynchronize (s[i]);
    }

    // Get the data of the final result.
    float** hostData = (float**)malloc (nDev * sizeof (float*));
    for (int i = 0; i < nDev; i++) {
        hostData[i] = (float*)malloc (size * sizeof (float));
    }
    for (int i = 0; i < nDev; i++) {
        hipMemcpy (hostData[i], recvbuff[i], size * sizeof (float), hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < nDev; i++) {
        printf ("[MPI Rank %d] Success: ", myRank * nDev + i);
        for (int j = 0; j < size; j++) {
            printf ("%f ", hostData[i][j]);
        }
        printf ("\n");
    }

    // Free all resources.
    for (int i = 0; i < nDev; i++) {
        ncclCommDestroy (comms[i]);
        CUDACHECK (hipFree (sendbuff[i]));
        CUDACHECK (hipFree (recvbuff[i]));
        free (hostData[i]);
    }
    free (sendbuff);
    free (recvbuff);
    free (hostData);

    // finalizing NCCL
    for (int i = 0; i < nDev; i++) {
        ncclCommDestroy (comms[i]);
    }

    // finalizing MPI
    MPICHECK (MPI_Finalize ());
    return;
}

int main (int argc, char* argv[]) {
    int myRank, nRanks, localRank = 0;

    // MPI initialization.
    MPICHECK (MPI_Init (&argc, &argv));
    MPICHECK (MPI_Comm_rank (MPI_COMM_WORLD, &myRank));
    MPICHECK (MPI_Comm_size (MPI_COMM_WORLD, &nRanks));

    printf ("MPI Initialized: myRank: %d, nRanks: %d, localRank: %d\n", myRank,
            nRanks, localRank);

    // localRank calculation based on hostname.
    uint64_t* hostHashs = (uint64_t*)malloc (sizeof (uint64_t) * nRanks);
    char hostname[1024];
    getHostName (hostname, 1024);
    hostHashs[myRank] = getHostHash (hostname);
    MPICHECK (MPI_Allgather (MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs,
                             sizeof (uint64_t), MPI_BYTE, MPI_COMM_WORLD));
    for (int p = 0; p < nRanks; p++) {
        if (p == myRank)
            break;
        if (hostHashs[p] == hostHashs[myRank])
            localRank++;
    }

    non_blocking_all_to_all (myRank, nRanks, localRank);
}